#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

#define N 5
#define  INF 99999

using namespace std;

__global__ void RoyFloyd(int matrixGraph[N][N])
{

	int i = threadIdx.x;
	int j = threadIdx.y;
	for (int k = 1; k <= N; k++)
	{
		if (matrixGraph[i][k] + matrixGraph[k][j] < matrixGraph[i][j]) {
			matrixGraph[i][j] = matrixGraph[i][k] + matrixGraph[k][j];
		}
	}
}

int h_matrixGraph[][] = {
	{ 0, 2, INF, 10, INF },
{ 2, 0, 3, INF, INF },
{ INF, 3, 0, 1, 8 },
{ 10, INF, 1, 0, INF },
{ INF, INF, 8, INF, 0 }
};

int main()
{
	int *d_matrix;

	hipMalloc(&d_matrix, N*N * sizeof(int)); //alocam memorie liniara

	for (int h_k = 1; h_k <= N; h_k++)
	{
		int* d_k;

		hipMemcpy(d_matrix, h_matrixGraph, N * N * sizeof(int), hipMemcpyHostToDevice); //transferam memoria din host in device
		hipMalloc(&d_k, sizeof(int)); //alocare memorie
		hipMemcpy(d_k, &h_k, sizeof(int), hipMemcpyHostToDevice); //transferam din host in device
		
		int numBlocks = 1;

		dim3 threadsPerBlock(N, N); //invocare kernel creem dimensiunea, mai exact doua dimensiuni pentru matrice
		RoyFloyd<<<numBlocks, threadsPerBlock>>>(d_matrix);
		hipMemcpy(h_matrixGraph, d_matrix, N * N * sizeof(int), hipMemcpyDeviceToHost);
	}

	for (int i = 1; i <= N; i++) {
		for (int j = 1; j <= N; j++)
		{
			if (h_matrixGraph[i][j] == INF)
				printf("-, ");
			else
				printf("%d, ", h_matrixGraph[i][j]);
		}
		printf("\n");
	}

	hipFree(h_matrixGraph); //eliberam memoria
	hipFree(d_matrix);
	system("pause");
	return 0;

}