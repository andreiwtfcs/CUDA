#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>

#define NR 5
#define  INF 99999

using namespace std;

int main()
{
	int *d_matrix;
	
	int h_matrixGraph[][] = {
		{ 0, 2, INF, 10, INF },
		{ 2, 0, 3, INF, INF },
		{ INF, 3, 0, 1, 8 },
		{ 10, INF, 1, 0, INF },
		{ INF, INF, 8, INF, 0 }
	};

	hipMalloc(&d_matrix, NR*NR * sizeof(int)); //alocam memorie liniara

	for (int h_k = 1; h_k <= NR; h_k++)
	{
		int* d_k;

		hipMemcpy(d_matrix, h_matrixGraph, NR * NR * sizeof(int), hipMemcpyHostToDevice); //transferam memoria din host in device
		hipMalloc(&d_k, sizeof(int)); //alocare memorie
		hipMemcpy(d_k, &h_k, sizeof(int), hipMemcpyHostToDevice); //transferam din host in device
		
		int numBlocks = 1;

		dim3 threadsPerBlock(NR, NR); //invocare kernel creem dimensiunea, mai exact doua dimensiuni pentru matrice
		RoyFloyd<<<numBlocks, threadsPerBlock>>>(d_matrix);
		hipMemcpy(h_matrixGraph, d_matrix, NR * NR * sizeof(int), hipMemcpyDeviceToHost);
	}

	for (int i = 1; i <= NR; i++) {
		for (int j = 1; j <= NR; j++)
		{
			if (h_matrixGraph[i][j] == INF)
				printf("-, ");
			else
				printf("%d, ", h_matrixGraph[i][j]);
		}
		printf("\n");
	}

	hipFree(h_matrixGraph); //eliberam memoria
	hipFree(d_matrix);
	system("pause");
	return 0;

}

__global__ void RoyFloyd(int matrixGraph[NR][NR])
{

	int i = threadIdx.x;
	int j = threadIdx.y;
	for (int k = 1; k <= NR; k++)
	{
		if (matrixGraph[i][k] + matrixGraph[k][j] < matrixGraph[i][j]) {
			matrixGraph[i][j] = matrixGraph[i][k] + matrixGraph[k][j];
		}
	}
}